#include "hip/hip_runtime.h"
/*
Highly Optimized Object-Oriented Molecular Dynamics (HOOMD) Open
Source Software License
Copyright (c) 2008 Ames Laboratory Iowa State University
All rights reserved.

Redistribution and use of HOOMD, in source and binary forms, with or
without modification, are permitted, provided that the following
conditions are met:

* Redistributions of source code must retain the above copyright notice,
this list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names HOOMD's
contributors may be used to endorse or promote products derived from this
software without specific prior written permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND
CONTRIBUTORS ``AS IS''  AND ANY EXPRESS OR IMPLIED WARRANTIES,
INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY
AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. 

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS  BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.
*/

// $Id: StochasticForceGPU.cu 1524 2008-12-04 23:03:55Z joaander $
// $URL: https://svn2.assembla.com/svn/hoomd/tags/hoomd-0.8.0/src/cuda/StochasticForceGPU.cu $

#include "StochasticForceGPU.cuh"
#include "gpu_settings.h"
#include "saruprngCUDA.h"

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif


/*! \file StochasticForceGPU.cu
	\brief Defines GPU kernel code for calculating the stochastic forces. Used by StochasticForceComputeGPU.
*/

//! Texture for reading particle velocities
texture<float4, 1, hipReadModeElementType> pdata_vel_tex;

//! Texture for reading particle positions
texture<float, 1, hipReadModeElementType> pdata_type_tex;

//! Texture for reading particle tags
texture<unsigned int, 1, hipReadModeElementType> pdata_tag_tex;

//! Kernel for calculating stochastic forces
/*! This kernel is called to apply stochastic heat bath forces to all N particles in conjunction with a Brownian Dynamics Simulations

	\param force_data Device memory array to write calculated forces to
	\param pdata Particle data on the GPU to calculate forces on
	\param dt Timestep of the simulation
	\param T Temperature of the bath
	\param d_gammas Gamma coefficients that govern the coupling of the particle to the bath.
	\param gamma_length length of the gamma array (number of particle types)
	\param seed Seed value that will be incoporated into the seed of the Saru RNG
	\param iteration current time step (hashed with other quantities to seed the RNG)
	
	\a gammas is a pointer to an array in memory. \c gamma[i] is \a gamma for the particle type \a i.
	The values in d_gammas are read into shared memory, so \c gamma_length*sizeof(float) bytes of extern 
	shared memory must be allocated for the kernel call.
	
	Developer information:
	Each block will calculate the forces on a block of particles.
	Each thread will calculate the total stochastic force on one particle.
	The RNG state vectors should permit a coalesced read, but this fact should be checked.
	
*/
extern "C" __global__ void gpu_compute_stochastic_forces_kernel(gpu_force_data_arrays force_data, gpu_pdata_arrays pdata, float dt, float T, float *d_gammas, int gamma_length, unsigned int seed, unsigned int iteration)
	{
	
	// read in the gammas (1 dimensional array)
	extern __shared__ float s_gammas[];
	for (int cur_offset = 0; cur_offset < gamma_length; cur_offset += blockDim.x)
		{
		if (cur_offset + threadIdx.x < gamma_length)
			s_gammas[cur_offset + threadIdx.x] = d_gammas[cur_offset + threadIdx.x];
		}
	__syncthreads();
	
	// start by identifying which particle we are to handle
	int idx_local = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx_local >= pdata.local_num)
		return;
	
	int idx_global = idx_local + pdata.local_beg;

	// read in the velocity of our particle. Texture reads of float4's are faster than global reads on compute 1.0 hardware
	// (MEM TRANSFER: 16 bytes)
	float4 vel = tex1Dfetch(pdata_vel_tex, idx_global);

	// read in the type of our particle. A texture read of only the fourth part of the position float4 (where type is stored) is used.  
	// (MEM TRANSFER: 4 bytes)
	float type_f = tex1Dfetch(pdata_type_tex, idx_global*4 + 3);
	int typ = __float_as_int(type_f);
	
	// read in the tag of our particle. 
	// (MEM TRANSFER: 4 bytes)
	unsigned int ptag = tex1Dfetch(pdata_tag_tex, idx_global);	
	
	// Calculate Coefficient of Friction
	//type = 0;   //May use this for benchmarking the impact of doing a second texture read just for particle type
	float coeff_fric = sqrtf(6.0f * s_gammas[typ] * T/ dt);
	
	// initialize the force to 0
	float4 force = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	
	//Initialize the Random Number Generator
	SaruGPU s(ptag, iteration, seed); // 3 dimensional seeding

	float randomx=s.f(-1.0, 1.0);
	float randomy=s.f(-1.0, 1.0);
	float randomz=s.f(-1.0, 1.0);

	// Generate random number and generate x, y, and z forces respectively
	force.x += randomx*coeff_fric - s_gammas[typ]*vel.x;
	force.y += randomy*coeff_fric - s_gammas[typ]*vel.y;
	force.z += randomz*coeff_fric - s_gammas[typ]*vel.z;

	// stochastic forces do not contribute to potential energy

	// now that the force calculation is complete, write out the result (MEM TRANSFER: 16 bytes)
	force_data.force[idx_local] = force;
	
	}


/*! \param force_data Force data on GPU to write forces to
	\param pdata Particle data on the GPU to perform the calculation on
	\param dt Timestep
	\param T Temperature values
	\param d_gammas The coefficients of friction for each particle type
	\param gamma_length  The length of d_gamma array
	\param iteration current time step (hashed with other quantities to seed the RNG)
	\param seed seed for the RNG to use in thread  (is hashed with other internal timestep depedent seeds)
	\param block_size Block size to execute
	
	\returns Any error code resulting from the kernel launch
	\note Always returns hipSuccess in release builds to avoid the hipDeviceSynchronize()
*/
hipError_t gpu_compute_stochastic_forces(const gpu_force_data_arrays& force_data, const gpu_pdata_arrays &pdata, float dt, float T, float *d_gammas, unsigned int seed, unsigned int iteration, int gamma_length, int block_size)
	{
	assert(d_gammas);
	assert(gamma_length > 0);

	// setup the grid to run the kernel
	dim3 grid( (int)ceil((double)pdata.local_num / (double)block_size), 1, 1);
	dim3 threads(block_size, 1, 1);

	// bind the velocity texture
	hipError_t error = hipBindTexture(0, pdata_vel_tex, pdata.vel, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;

	// bind the position texture  (this is done only to retrieve the particle type)
	error = hipBindTexture(0, pdata_type_tex, pdata.pos, sizeof(float4) * pdata.N);
	if (error != hipSuccess)
		return error;
	
	// bind the tag texture
	error = hipBindTexture(0, pdata_tag_tex, pdata.tag, sizeof(unsigned int) * pdata.N);
	if (error != hipSuccess)
		return error;
		
    // run the kernel
    gpu_compute_stochastic_forces_kernel<<< grid, threads, sizeof(float)*gamma_length>>>(force_data, pdata, dt, T, d_gammas, gamma_length, seed, iteration);

	if (!g_gpu_error_checking)
		{
		return hipSuccess;
		}
	else
		{
		hipDeviceSynchronize();
		return hipGetLastError();
		}
	}

// vim:syntax=cpp
